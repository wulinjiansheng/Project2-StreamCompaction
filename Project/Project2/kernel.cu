#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h> 
#include "CPUcode.h"
#include <time.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/count.h>

#define blockSize 256
bool print = false;

const int originN = 1000;

//Part2
__global__ void NaivePrefixSum(int *in,int *out,int n,int d)
{
	int k = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(k<n)
	{
		if(k>= (int)pow(2.0,d-1))
		    out[k] = in[k-(int)pow(2.0,d-1)] + in[k];
	    else
		    out[k] = in[k];	    
	}	
}

void InitNaivePrefixSum(int *origin,int *result,int N)
{
	float time;
	int *in;
	int *out;
	int* temp;
	hipMalloc((void**)&in,sizeof(int)*N);
	hipMalloc((void**)&out,sizeof(int)*N);
	hipMemcpy(in, origin, sizeof(int)*N, hipMemcpyHostToDevice);	
	int numBlocks = (int)ceil(N/(float)blockSize);
	double t = gettime();
	for(int d=1;(int)pow(2.0,d-1)<=N;d++)
	{
        NaivePrefixSum<<<numBlocks,blockSize>>> (in,out,N,d);
		//update indata
		temp = in;
        in=out;
		out = temp;
	}
	out = in;
	t = gettime()-t;
	cout<<"  Run time:"<<1000*t<<" ms"<<endl;

	result[0] = 0;
	hipMemcpy(result+1, out, sizeof(int)*N, hipMemcpyDeviceToHost); 
	hipFree(in);
	hipFree(out);
}

//Part3 a
__global__ void PrefixSumSharedM(int  *in,int  *out,int n)
{
	int k = threadIdx.x;
	__shared__ int s_in [blockSize];
	__shared__ int s_out[blockSize];

	s_in[k] = in[k];
	__syncthreads();

	for(int d=1;(int)pow(2.0,d-1)<=n;d++)
	{	
		if(k>= (int)pow(2.0,d-1))
			s_out[k] = s_in[k-(int)pow(2.0,d-1)] + s_in[k];
		else
			s_out[k] = s_in[k];	    

		s_in[k] = s_out[k];
		__syncthreads();
	}

	out[k] = s_out[k];
}

void InitPrefixSumSharedM(int *origin,int *result,int N)
{
	float time;
	int *in,*out;
	hipMalloc((void**)&in,sizeof(int)*N);
	hipMalloc((void**)&out,sizeof(int)*N);
	hipMemcpy(in, origin, sizeof(int)*N, hipMemcpyHostToDevice);
	int numBlocks = (int)ceil(N/(float)blockSize);
	if(numBlocks>1)
	{
		cout<<"  Error, more numbers than blocksize."<<endl;
		return;
	}

	double t = gettime();
    PrefixSumSharedM<<<numBlocks,blockSize>>> (in,out,N);
	t = gettime() - t;
	cout<<"  Run time:"<<1000*t<<" ms"<<endl;
	result[0] = 0;
	hipMemcpy(result+1, out, sizeof(int)*N, hipMemcpyDeviceToHost); 
	hipFree(in);
	hipFree(out);
}

//Part3 b
__global__ void OPPrefixSumSharedM(int  *in,int  *out,int n,int *sums)
{
	int gk = blockDim.x * blockIdx.x + threadIdx.x;
	if(gk>=n)
		return;

	int k = threadIdx.x;
	__shared__ int s_in [blockSize];
	__shared__ int s_out[blockSize];

	s_in[k] = in[gk];
	__syncthreads();

	for(int d=1;(int)pow(2.0,d-1)<=blockSize;d++)
	{	
		if(k>= (int)pow(2.0,d-1))
			s_out[k] = s_in[k-(int)pow(2.0,d-1)] + s_in[k];
		else
			s_out[k] = s_in[k];	    

		s_in[k] = s_out[k];

		__syncthreads();
	}

	if(k==blockSize-1)
	   sums[blockIdx.x] = s_out[k]; 
	out[gk] = s_out[k]; 
}

__global__ void AddInc(int  *in,int *incs)
{
	int gk = blockDim.x * blockIdx.x + threadIdx.x;
	if(blockIdx.x>=1)
	    in[gk] += incs[blockIdx.x-1];
}

void InitOPPrefixSumSharedM(int *origin,int *result,int N)
{
	float time;
	int *in,*out,*sums,*incr;
	int numBlocks = (int)ceil(N/(float)blockSize);
	hipMalloc((void**)&in,sizeof(int)*N);
	hipMalloc((void**)&out,sizeof(int)*N);
	hipMalloc((void**)&sums,sizeof(int)*numBlocks);
	hipMalloc((void**)&incr,sizeof(int)*numBlocks);
	hipMemcpy(in, origin, sizeof(int)*N, hipMemcpyHostToDevice);

	double t = gettime();
	//Get sums for each block
	OPPrefixSumSharedM<<<numBlocks,blockSize>>> (in,out,N,sums);

	if(numBlocks>1)
	{
		//Get incrs
		int *temp;
		int numBlocks2 = (int)ceil(numBlocks/(float)blockSize);
		for(int d=1;(int)pow(2.0,d-1)<=numBlocks;d++)
		{
			NaivePrefixSum<<<numBlocks2,blockSize>>> (sums,incr,numBlocks,d);
			//update indata
			temp = sums;
			sums = incr;
			incr = temp;
		}
		incr = sums;

		//Add to out
		AddInc<<<numBlocks,blockSize>>> (out,incr);
	}	

	t = gettime()-t;
	cout<<"  Run time:"<<1000*t<<" ms"<<endl;
	result[0] = 0;
	hipMemcpy(result+1, out, sizeof(int)*(N-1), hipMemcpyDeviceToHost); 

	hipFree(in);
	hipFree(out);
	hipFree(sums);
	hipFree(incr);
}

//Part 4
__global__ void GPUScatter(int *in,int *newin,bool *inbool,int n)
{
	int gk = blockDim.x * blockIdx.x + threadIdx.x;
	if(gk>=n)
		return;

	if(in[gk]>0)
	{
		inbool[gk] = true;
		newin[gk] = 1;
	}
	else if(in[gk]==0)
	{
		inbool[gk] = false;
		newin[gk] = 0;
	}
}

__global__ void GetMaxIndex(int *in,int *maxindex,int n)
{
	int gk = blockDim.x * blockIdx.x + threadIdx.x;
	if(gk>=n)
		return;

	if(gk==n-1)
	    maxindex[0] = in[gk];
}

__global__ void StreamCompact(int *in,int *out,bool *inbool,int *final,int n)
{
	int gk = blockDim.x * blockIdx.x + threadIdx.x;
	if(gk>=n)
		return;

	if(out[gk]>0&&inbool[gk]==true)
		final[out[gk]-1] = in[gk];
}

int* InitStreamCompact(int *origin,int N,int &l)
{
	float time;
	int *in,*newin,*out,*final,*sums,*incr,*maxindex;
	bool *inbool;
	int numBlocks = (int)ceil(N/(float)blockSize);
	hipMalloc((void**)&maxindex,sizeof(int)*1);
	hipMalloc((void**)&inbool,sizeof(bool)*N);
	hipMalloc((void**)&in,sizeof(int)*N);
	hipMalloc((void**)&newin,sizeof(int)*N);
	hipMalloc((void**)&out,sizeof(int)*N);
	hipMalloc((void**)&sums,sizeof(int)*numBlocks);
	hipMalloc((void**)&incr,sizeof(int)*numBlocks);
	hipMemcpy(in, origin, sizeof(int)*N, hipMemcpyHostToDevice);
	
	double t = gettime();
	//Scatter
	GPUScatter<<<numBlocks,blockSize>>> (in,newin,inbool,N);

	//Scan
	//Get sums for each block
	OPPrefixSumSharedM<<<numBlocks,blockSize>>> (newin,out,N,sums);

	if(numBlocks>1)
	{
		//Get incrs
		int *temp;
		int numBlocks2 = (int)ceil(numBlocks/(float)blockSize);
		for(int d=1;(int)pow(2.0,d-1)<=numBlocks;d++)
		{
			NaivePrefixSum<<<numBlocks2,blockSize>>> (sums,incr,numBlocks,d);
			//update indata
			temp = sums;
			sums = incr;
			incr = temp;
		}
		incr = sums;

		//Add to out
		AddInc<<<numBlocks,blockSize>>> (out,incr);
	}	

	//Get Maxindex for out array
	GetMaxIndex<<<numBlocks,blockSize>>> (out,maxindex,N);
	int max;
	hipMemcpy(&max, maxindex, sizeof(int)*1, hipMemcpyDeviceToHost); 


	//Generate final result
	hipMalloc((void**)&final,sizeof(int)*max);
	StreamCompact<<<numBlocks,blockSize>>> (in,out,inbool,final,N);

	t = gettime()-t;
	cout<<"  Run time:"<<1000*t<<" ms"<<endl;
	int *result = new int[max];
	hipMemcpy(result, final, sizeof(int)*max, hipMemcpyDeviceToHost); 
	hipFree(in);
	hipFree(out);
	hipFree(sums);
	hipFree(incr);
	hipFree(final);
	l = max;
	return result;
}


//Thrust
struct not_Zero
{
	__host__ __device__
		bool operator()(const int x)
	{
		if(x>0) return true;
		else return false;
	}
};


int* ThrustStreamCompact(int *origin,int N,int &l)
{
	//Count how many numbers is not 0
	int finallength = thrust::count_if(origin, origin+N,not_Zero());
	int *result = new int[finallength];
	thrust::copy_if(origin, origin+N,result,not_Zero());
	l = finallength;

	return result;
}


bool Verify(int *r1,int*r2,int l)
{
	for(int i=0;i<l;i++)
	{
		if(r1[i]!=r2[i])
		{
			cout<<"Wrong result at index"<<i<<endl;
			cout<<endl;
			return false;
		}
	}
	cout<<"Virified!"<<endl;
	cout<<endl;
	return true;
}

void main()
{
	int *origin;
	origin = new int[originN];
	for(int i=0;i<originN;i++)
		origin[i] =  rand()%10;
	if(print)
	{
		cout<<"Array:"<<endl;
		for(int i=0;i<originN;i++)
			cout<<origin[i]<<" ";
	}


	cout<<endl;


	//Part1
	cout<<"Serial Version PrefixSum:"<<endl;
	int* result = new int[originN];
	double t = gettime();
	PrefixSum(origin,result,originN);
	t = gettime() - t;
	cout<<"  Run time:"<<1000*t<<" ms"<<endl;
	cout<<endl;


	//Part2
	cout<<"Naive GPU Version PrefixSum:"<<endl;
	int* result2 = new int[originN+1];
	InitNaivePrefixSum(origin,result2,originN);
	//Verify(result,result2,originN);
	cout<<endl;

	//Part3 a
	cout<<"GPU Version PrefixSum with Shared Memory(Single Block):"<<endl;
	int* result3 = new int[originN+1];
	InitPrefixSumSharedM(origin,result3,originN);
	//Verify(result,result3,originN);
	cout<<endl;

	//Part3 b
	cout<<"GPU Version PrefixSum with Shared Memory(Arbitrary length):"<<endl;
	int* result4 = new int[originN];
	InitOPPrefixSumSharedM(origin,result4,originN);
	//Verify(result,result4,originN);
	cout<<endl;


	//Part4 
	int l = 0; //Will be the length of compact array
	cout<<"Serial Version StreamCompact:"<<endl;
	t = gettime();
	int* test =	StreamCompact(origin,originN,l);
	t = gettime() - t;
	cout<<"  Run time:"<<1000*t<<" ms"<<endl;
	cout<<endl;


	cout<<"GPU version Stream Compact:"<<endl;
	int* result5 = InitStreamCompact(origin,originN,l);
	//Verify(test,result5,l);
	cout<<endl;


	cout<<"Thrust version Stream Compact:"<<endl;
	t = gettime();
	int* result6 = ThrustStreamCompact(origin,originN,l);
	t = gettime() - t;
	cout<<"  Run time:"<<1000*t<<" ms"<<endl;
	//Verify(test,result6,l);
	cout<<endl;

	delete []origin;
	delete []result;
	delete []result2;
	delete []result3;
	delete []result4;
	delete []result5;
	delete []result6;

	return;
}